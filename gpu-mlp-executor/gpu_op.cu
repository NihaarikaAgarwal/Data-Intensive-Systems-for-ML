#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

__global__ void array_set_kernel(index_t n, float *data, float value) {
  index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n)
    data[idx] = value;
}

int DLGpuArraySet(DLArrayHandle arr, float value) { /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < arr->ndim; i++)
    n *= arr->shape[i];

  float *data = (float *) arr->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  array_set_kernel<<<n_blocks, thread_per_block>>>(n, data, value);
  return 0;
}

__global__ void broadcast_to_kernel(const float *input_data, 
  float *output_data,
  index_t input_n,
  index_t output_n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < output_n) {
output_data[idx] = input_data[idx % input_n];
}
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  index_t input_n = 1;
  for (int i = 0; i < input->ndim; i++)
    input_n *= input->shape[i];

  index_t output_n = 1;
  for (int i = 0; i < output->ndim; i++)
    output_n *= output->shape[i];

  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;

  int thread_per_block = 512;
  int n_blocks = (output_n + thread_per_block - 1) / thread_per_block;
  broadcast_to_kernel<<<n_blocks, thread_per_block>>>(input_data, output_data,
                                                      input_n, output_n);
  return 0;
}

__global__ void reduce_sum_axis_zero_kernel(const float *input_data, 
  float *output_data,
  index_t input_n,
  index_t output_n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < output_n) {
output_data[idx] = 0.0; 
for (index_t i = 0; i < input_n / output_n; i++)
output_data[idx] += input_data[i * output_n + idx];
}
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  index_t input_n = 1;
  for (int i = 0; i < input->ndim; i++)
    input_n *= input->shape[i];

  index_t output_n = 1;
  for (int i = 0; i < output->ndim; i++)
    output_n *= output->shape[i];

  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;

  int thread_per_block = 512;
  int n_blocks = (output_n + thread_per_block - 1) / thread_per_block;
  reduce_sum_axis_zero_kernel<<<n_blocks, thread_per_block>>>(input_data, 
                                                              output_data,
                                                              input_n, 
                                                              output_n);
  return 0;
}

__global__ void matrix_elementwise_add_kernel(const float *matA_data,
  const float *matB_data,
  float *output_data, 
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = matA_data[idx] + matB_data[idx];
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < output->ndim; i++)
    n *= output->shape[i];

  const float *matA_data = (const float *) matA->data;
  const float *matB_data = (const float *) matB->data;
  float *output_data = (float *) output->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  matrix_elementwise_add_kernel<<<n_blocks, thread_per_block>>>(matA_data, 
                                                                matB_data, 
                                                                output_data,
                                                                n); 
  return 0;
}

__global__ void matrix_elementwise_add_by_const_kernel(const float *input_data,
  float *output_data,
  float val,
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = input_data[idx] + val;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < output->ndim; i++)
    n *= output->shape[i];

  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  matrix_elementwise_add_by_const_kernel<<<n_blocks, thread_per_block>>>(input_data, 
                                                                         output_data,
                                                                         val,
                                                                         n); 
  return 0;
}

__global__ void matrix_elementwise_multiply_kernel(const float *matA_data,
  const float *matB_data,
  float *output_data, 
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = matA_data[idx] * matB_data[idx];
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < output->ndim; i++)
    n *= output->shape[i];

  const float *matA_data = (const float *) matA->data;
  const float *matB_data = (const float *) matB->data;
  float *output_data = (float *) output->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  matrix_elementwise_multiply_kernel<<<n_blocks, thread_per_block>>>(matA_data, 
                                                                     matB_data, 
                                                                     output_data,
                                                                     n);
  return 0;
}

__global__ void matrix_elementwise_multiply_by_const_kernel(const float *input_data,
  float *output_data,
  float val,
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = input_data[idx] * val;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < output->ndim; i++)
    n *= output->shape[i];

  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  matrix_elementwise_multiply_by_const_kernel<<<n_blocks, thread_per_block>>>(input_data, 
                                                                              output_data,
                                                                              val,
                                                                              n); 
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  /* TODO: Your code here */
  // Hint: use cublas
  // cublas assume matrix is column major
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    printf("CUBLAS initialization failed\n");

  const float *matA_data = (const float *) matA->data;
  const float *matB_data = (const float *) matB->data;
  float *matC_data = (float *) matC->data;

  hipblasOperation_t transa = transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb = transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  int m = transposeB ? matB->shape[0] : matB->shape[1];
  int n = transposeA ? matA->shape[1] : matA->shape[0];
  int k = transposeA ? matA->shape[0] : matA->shape[1];

  float alpha = 1.0f;
  float beta = 0.0f;
  stat = hipblasSgemm(handle, transb, transa,
                     m, n, k,
                     &alpha, matB_data, matB->shape[1],
                     matA_data, matA->shape[1], 
                     &beta, matC_data, m);
  if (stat != HIPBLAS_STATUS_SUCCESS) 
    printf("CUBLAS kernel execution error.\n");
                 
  stat = hipblasDestroy(handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) 
      printf("CUBLAS shutdown error\n");
                  
  return 0;
}

__global__ void relu_kernel(const float *input_data, float *output_data, 
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = input_data[idx] > 0 ? input_data[idx] : 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < output->ndim; i++)
    n *= output->shape[i];

  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;
  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  relu_kernel<<<n_blocks, thread_per_block>>>(input_data, output_data, n);
  return 0;
}

__global__ void relu_gradient_kernel(const float *input_data, 
  const float *in_grad_data, 
  float *output_data,
  index_t n) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
if (idx < n) 
output_data[idx] = input_data[idx] > 0 ? in_grad_data[idx] : 0;
}


int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  /* TODO: Your code here */
  index_t n = 1;
  for (int i = 0; i < input->ndim; i++)
    n *= input->shape[i];

  const float *input_data = (const float *) input->data;
  const float *in_grad_data = (const float *) in_grad->data;
  float *output_data = (float *) output->data;

  int thread_per_block = 512;
  int n_blocks = (n + thread_per_block - 1) / thread_per_block;
  relu_gradient_kernel<<<n_blocks, thread_per_block>>>(input_data, 
                                                       in_grad_data,
                                                       output_data,
                                                       n);
  return 0;
}

__global__ void softmax_kernel(const float *input_data, float *output_data,
  index_t row, index_t col) {
index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
// Operate on each row
if (idx < row) {
// Find max
float max_val = input_data[idx*col];
for (int i = idx*col; i < (idx+1) * col; i++) 
max_val = max(max_val, input_data[i]);

// Subtract max and exp
float sum = 0.0;
for (int i = idx*col; i < (idx+1) * col; i++) {
output_data[i] = exp(input_data[i] - max_val);
sum += output_data[i];
}

// Divide sum of this row
for (int i = idx*col; i < (idx+1) * col; i++) 
output_data[i] /= sum;
}

}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  const float *input_data = (const float *) input->data;
  float *output_data = (float *) output->data;

  int n_blocks = output->shape[0];
  softmax_kernel<<<n_blocks, 1>>>(input_data, output_data, output->shape[0],
                                  output->shape[1]);
  return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}
